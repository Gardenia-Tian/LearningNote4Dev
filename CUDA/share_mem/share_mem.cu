#include<hip/hip_runtime.h>
#include<bits/stdc++.h>

#define N 128 // 数组大小

__global__ void smem(uint32_t *a) {
  __shared__ uint32_t smem[128];
  uint32_t tid = threadIdx.x;
  for (int i = 0; i < 4; i++) {
    smem[i * 32 + tid] = tid;
  }
  __syncthreads();
  a[tid] =smem[tid];
}

__global__ void smem_3(uint32_t *a) {
  __shared__ uint32_t smem[128];
  uint32_t tid = threadIdx.x;
  for (int i = 0; i < 4; i++) {
    smem[i * 32 + tid] = tid;
  }
  __syncthreads();
  reinterpret_cast<uint2 *>(a)[tid] =
      reinterpret_cast<const uint2 *>(smem)[tid/2];
}


int main() {
    // 在主机上声明一个数组
    uint32_t a[N];

    // 在设备上声明一个数组
    uint32_t *d_a;
    hipMalloc((void**)&d_a, N * sizeof(uint32_t));

    // 调用 CUDA 核函数
    // smem<<<1, 32>>>(d_a);
    smem_3<<<1, 32>>>(d_a);

    // 将结果从设备复制回主机
    hipMemcpy(a, d_a, N * sizeof(uint32_t), hipMemcpyDeviceToHost);

    // 打印结果
    printf("Result:\n");
    for (int i = 0; i < 4; i++) {
        for(int j = 0;j<32;j++){
            printf("%d ", a[i * 32 + j]);
        }
        printf("\n");
    }
    printf("\n");

    // 释放设备上的数组
    hipFree(d_a);

    return 0;
}
