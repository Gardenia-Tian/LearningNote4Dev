#include<hip/hip_runtime.h>
#include<bits/stdc++.h>


#define WARP_SIZE 32
#define INT4(value) (reinterpret_cast<int4*>(&(value))[0])


// smem + tile
// A : M * K   B: K * N  C: M * N
__global__ void sgemm(float* a, float* b, float* c, int M, int N, int K){
    // constexpr 表示常量语义
    // const 表示只读语义
    constexpr int BM = 32;
    constexpr int BN = 32;
    constexpr int BK = 32;
    __shared__ float s_a[BM][BK];
    __shared__ float s_b[BK][BN];

    int ty = threadIdx.y, tx = threadIdx.x;
    int by = blockIdx.y, bx = threadIdx.x; 
    // 在结果矩阵C中, 所计算的单元的位置
    int row = by * BM + ty;
    int col = bx * BN + tx;

    float tempVal = 0;

    for(int ph = 0; ph < (K + BK - 1)/ BK ;ph++){
        if(row < M && ph * BK + tx < K){
            s_a[ty][tx] = a[row * K + (ph * BK + tx)];
        }else{
            s_a[ty][tx] = 0;
        }
        if(ph * BK + ty<K && col < N){
            s_b[ty][tx] = b[(ph * BK + ty) * M + col];
        }else{
            s_b[ty][tx] = 0;
        }
        __syncthreads();

        for(int k = 0;k < BK ;k++){
            tempVal += s_a[ty][k] * s_b[k][tx];
        }
        __syncthreads();
    }
    if(row < M && col < N){
        c[row * N + col] = tempVal;
    }

}



