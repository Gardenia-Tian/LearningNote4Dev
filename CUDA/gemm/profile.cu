#include "gemm.cuh"



int main(int argc, char* argv[]) {
    // constexpr int M = 40960;
    // constexpr int K = 20480;
    // constexpr int N = 40960;
    int M = atoi(argv[1]);
    int K = atoi(argv[2]);
    int N = atoi(argv[3]);

    // float* a = (float*)malloc(sizeof(float) * M * K);
    // float* b = (float*)malloc(sizeof(float) * K * N);
    float* c = (float*)malloc(sizeof(float) * M * N);
    float* test = (float*)malloc(sizeof(float) * M * N);

    float* da, * db, * dc;
    float* dtest;
    hipMalloc((void**)&da, sizeof(float) * M * K);
    hipMalloc((void**)&db, sizeof(float) * K * N);
    hipMalloc((void**)&dc, sizeof(float) * M * N);
    hipMalloc((void**)&dtest, sizeof(float) * M * N);

    dim3 blockSize(32, 32);
    dim3 gridSize((K + blockSize.x - 1) / blockSize.x, (M + blockSize.y - 1) / blockSize.y);

    randomInit<float> << <gridSize, blockSize >> > (da, M, K, time(NULL));

    gridSize.x = (N + blockSize.x - 1) / blockSize.x;
    gridSize.y = (K + blockSize.y - 1) / blockSize.y;
    randomInit<float> << <gridSize, blockSize >> > (db, K, N, time(NULL));

    gridSize.x = (N + blockSize.x - 1) / blockSize.x;
    gridSize.y = (M + blockSize.y - 1) / blockSize.y;


    gemm_v0<float> << <gridSize, blockSize >> > (da, db, dtest, M, N, K);
    // gemm_v1<float><<<gridSize, blockSize>>>(da, db, dtest, M, N, K);
    // gemm_shared_tile<32, 32, 32, float> << <gridSize, blockSize >> > (da, db, dc, M, N, K);
    gemm_shared_transposeA_tile<32, 32, 32, float> << <gridSize, blockSize >> > (da, db, dc, M, N, K);

    hipMemcpy(c, dc, sizeof(float) * M * N, hipMemcpyDeviceToHost);
    hipMemcpy(test, dtest, sizeof(float) * M * N, hipMemcpyDeviceToHost);


    hipDeviceSynchronize();

    bool resRight = checkVal(c, test, M*N);
    printf("%d\n", resRight);


    free(c);
    free(test);
    hipFree(da);
    hipFree(db);
    hipFree(dc);
    hipFree(dtest);

    return 0;
}
