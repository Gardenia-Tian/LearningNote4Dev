#include "gemm.cuh"



int main(){
    constexpr int M = 4;
    constexpr int K = 2;
    constexpr int N = 4;

    // float* a = (float*)malloc(sizeof(float) * M * K);
    // float* b = (float*)malloc(sizeof(float) * K * N);
    float* c = (float*)malloc(sizeof(float) * M * N);
    float* test = (float*)malloc(sizeof(float) * M * N);

    float *da, *db, *dc;
    float *dtest;
    hipMalloc((void**)&da, sizeof(float) * M * K);
    hipMalloc((void**)&db, sizeof(float) * K * N);
    hipMalloc((void**)&dc, sizeof(float) * M * N);
    hipMalloc((void**)&dtest, sizeof(float) * M * N);
    
    dim3 blockSize(32, 32);
    dim3 gridSize((K + blockSize.x - 1) / blockSize.x, (M + blockSize.y - 1) / blockSize.y);

    randomInit<float><<<gridSize, blockSize>>>(da, M, K, time(NULL));

    gridSize.x = (N + blockSize.x - 1) / blockSize.x;
    gridSize.y = (K + blockSize.y - 1) / blockSize.y;
    randomInit<float><<<gridSize, blockSize>>>(db, K, N, time(NULL));

    gridSize.x = (N + blockSize.x - 1) / blockSize.x;
    gridSize.y = (M + blockSize.y - 1) / blockSize.y;

    // sgemm_v1<<<numBlocks, threadsPerBlock>>>(da, db, dc, M, N, K);
    
    gemm_v0<float><<<gridSize, blockSize>>>(da, db, dtest, M, N, K);


    hipMemcpy(c, dc, sizeof(float) * M * N, hipMemcpyDeviceToHost);
    hipMemcpy(test, dtest, sizeof(float) * M * N, hipMemcpyDeviceToHost);
 

    hipDeviceSynchronize();

    // bool resRight = checkVal(c, test, M*N);
    // printf("%d\n", resRight);


    free(c);
    free(test);
    hipFree(da);
    hipFree(db);
    hipFree(dc);
    hipFree(dtest);

    return 0;
}
