#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<bits/stdc++.h>

#define WARP_SIZE 32
#define INT4(value) (reinterpret_cast<int4*>(&(value))[0])


// CUDA 核函数，执行矩阵乘法
__global__ void gemm_v0(float *a, float *b, float *c,  int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;

    if (row < N && col < N) {
        for (int i = 0; i < K; ++i) {
            sum += a[row * K + i] * b[i * N + col];
        }
        c[row * N + col] = sum;
    }
}


// smem + tile
// A : M * K   B: K * N  C: M * N
__global__ void sgemm_v1(float* a, float* b, float* c, int M, int N, int K){
    // constexpr 表示常量语义
    // const 表示只读语义
    // 定义每一块的大小, 相当于tile_width
    constexpr int BM = 32;
    constexpr int BN = 32;
    constexpr int BK = 32;
    __shared__ float s_a[BM][BK];
    __shared__ float s_b[BK][BN];

    int ty = threadIdx.y, tx = threadIdx.x;
    int by = blockIdx.y, bx = threadIdx.x; 
    // 在结果矩阵C中, 所计算的单元的位置
    int row = by * BM + ty;
    int col = bx * BN + tx;

    float tempVal = 0;

    for(int ph = 0; ph < (K + BK - 1)/ BK ;ph++){
        if(row < M && ph * BK + tx < K){
            s_a[ty][tx] = a[row * K + (ph * BK + tx)];
        }else{
            s_a[ty][tx] = 0;
        }
        if(ph * BK + ty<K && col < N){
            s_b[ty][tx] = b[(ph * BK + ty) * M + col];
        }else{
            s_b[ty][tx] = 0;
        }
        __syncthreads();

        for(int k = 0;k < BK ;k++){
            tempVal += s_a[ty][k] * s_b[k][tx];
        }
        __syncthreads();
    }
    if(row < M && col < N){
        c[row * N + col] = tempVal;
    }
}


/*
shared memo 大幅度提升访存效率, 进而提高性能, 但是shared memory会存在band conflict
现象, 如果线程访问的数据在同一个bank内, 就要串行访问了.
在A*B的过程中, 每个线程访问矩阵A的一行, 所有的线程访问的是矩阵的A的列. 
而矩阵A在shared memory中是按行存储的.
同一个warp的不同线程的同一条load指令的访存地址是被间隔开的, 存在band conflict
优化点:
1. 矩阵A的shared memory按列存储
2. 让矩阵A的load数据, 使用一个load指令就可以完成
*/

/*
通过向量化内存访问提高性能:
a的share mem按列存储
*/
__global__ void sgemm_v2(float* a, float* b, float* c, int M, int N, int K){
    constexpr int BM = 32;
    constexpr int BK = 32;
    constexpr int BN = 32;
    // a -> BM*BK
    // b -> BK*BN
    // c -> BM*BN

    __shared__ float sa[BK][BM];
    __shared__ float sb[BK][BN];


    int tx = threadIdx.x, ty = threadIdx.y;
    int bx = blockIdx.x, by = blockIdx.y;
    int row = by * BM + ty;
    int col = bx * BN + tx; 
    int tid = ty * blockDim.x + tx;

    float tempVal = 0;

    for(int ph = 0; ph < (BK + K - 1) / BK ; ph++){
        reinterpret_cast<float4*>(sa)[tx][ty] = reinterpret_cast<float4*>(a)[row * K + ph * BK + tx];
        sb[ty][tx] = b[(ph * BK + ty) * K + col];
        __syncthreads();
    }

}

bool checkVal(float* a, float* b, int m){
    for(int i = 0;i<m;i++){
        if(a[i] != b[i]) return false;
    }
    return true;
}



int main(){
    constexpr int M = 2048;
    constexpr int K = 1024;
    constexpr int N = 2048;
    float* a = (float*)malloc(sizeof(float) * M * K);
    float* b = (float*)malloc(sizeof(float) * K * N);
    float* c = (float*)malloc(sizeof(float) * M * N);
    float* test = (float*)malloc(sizeof(float) * M * N);
    float *da, *db, *dc;
    float *dtest;
    hipMalloc((void**)&da, sizeof(float) * M * K);
    hipMalloc((void**)&db, sizeof(float) * K * N);
    hipMalloc((void**)&dc, sizeof(float) * M * N);
    hipMalloc((void**)&dtest, sizeof(float) * M * N);
    
    hipMemcpy(da, a, sizeof(float) * M * K, hipMemcpyHostToDevice);
    hipMemcpy(db, b, sizeof(float) * K * N, hipMemcpyHostToDevice);
    

    
    dim3 threadsPerBlock(32,32);
    dim3 numBlocks(M/32, N/32);
    
    
    


    sgemm_v1<<<numBlocks, threadsPerBlock>>>(da, db, dc, M, N, K);
    gemm_v0<<<numBlocks, threadsPerBlock>>>(da, db, dtest, M, N, K);
    
    hipMemcpy(c, dc, sizeof(float) * M * N, hipMemcpyDeviceToHost);
    hipMemcpy(test, dtest, sizeof(float) * M * N, hipMemcpyDeviceToHost);

    bool resRight = checkVal(c, test, M*N);
    printf("%d\n", resRight);

    free(a);
    free(b);
    free(c);
    free(test);
    hipFree(da);
    hipFree(db);
    hipFree(dc);
    hipFree(dtest);

    return 0;
}